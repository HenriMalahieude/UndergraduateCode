
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y; 

    if (row < dim && col < dim){
        C[row*dim + col] = A[row*dim + col] + B[row*dim + col];
    }

    /*************************************************************************/

}

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------
    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    const unsigned int g = ceil(dim / (float)BLOCK_SIZE);
    dim3 Gridder(g, g, 1);
    dim3 Blocker(BLOCK_SIZE, BLOCK_SIZE, 1);
    /*************************************************************************/
	
	// Invoke CUDA kernel -----------------------------------------------------
    matAdd<<<Gridder, Blocker>>>(dim, A, B, C);

    /*************************************************************************/
    //INSERT CODE HERE
	
    /*************************************************************************/

}

